#include "hip/hip_runtime.h"

#include <stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <windows.h>
#include <>
#include<hipblas.h>
#include<hipsparse.h>

#include<stdlib.h>
#include<time.h>



#define R_SIZE BLOCK_NUM * THREAD_NUM
#define M_SIZE R_SIZE * R_SIZE

#define FILE_BUFFER_LENGTH 30000









void get_rand_sparse(int m, int n, float** A, int percent)
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            A[i][j] = 0.0;

    int nnzNum = n * m * percent / 100;
    int* row = (int*)malloc(sizeof(int) * nnzNum);
    int* clo = (int*)malloc(sizeof(int) * nnzNum);
    float* value = (float*)malloc(sizeof(float) * nnzNum);


    //��������Ԫ

    srand((unsigned)time(NULL) + 23);
    for (int i = 0; i < nnzNum; i++)
    {
        row[i] = rand() % m;
    }

    srand((unsigned)time(NULL) + 43);
    for (int i = 0; i < nnzNum; i++)
    {
        clo[i] = rand() % n;
    }

    srand((unsigned)time(NULL) + 67);
    for (int i = 0; i < nnzNum; i++)
    {
        value[i] = (rand() % 10000) / 100.0 + 1.0;
    }

    for (int i = 0; i < nnzNum; i++)
    {
        A[row[i]][clo[i]] = value[i];



    }

    int nnz = 0;

    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
        {
            if (A[i][j] != 0)
                nnz++;
        }
    float per = nnz / (float)(m * n);
    printf("����ϡ���Ϊ%f�ľ���\n", per);

}



void crsmv(float** A, float* A_first_row)
{
    float* clo_B1_dim1 = (float*)malloc(sizeof(float) * 784 * 300);
    for (int i = 0; i < 784; i++)
        for (int j = 0; j < 300; j++)
        {
            clo_B1_dim1[i * 300 + j] = A[j][i];


        }//300*784 ��չ��


    int* nnzperrow = (int*)malloc(sizeof(int) * 300);

    int* GPU_nnzperrow;

    int nnztotal;

    float* GPU_cusparse_B;
    hipMalloc((void**)&GPU_cusparse_B, sizeof(float) * 300 * 784);
    hipMalloc((void**)&GPU_nnzperrow, sizeof(int) * 300);

    hipsparseHandle_t     handle = 0;//�������
    hipsparseMatDescr_t matb = 0;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&matb);
    hipsparseSetMatType(matb, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matb, HIPSPARSE_INDEX_BASE_ZERO);


    hipMemcpy(GPU_cusparse_B, clo_B1_dim1, sizeof(float) * 300 * 784, hipMemcpyHostToDevice);

    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, 300, 784, matb, GPU_cusparse_B, 300, GPU_nnzperrow, &nnztotal);//����ÿһ�з���Ԫ

    //printf("csr_nnnz:%d\n",nnztotal);
    hipMemcpy(nnzperrow, GPU_nnzperrow, sizeof(int) * 300, hipMemcpyDeviceToHost);

    //printf("NNZ ������%d ,", nnztotal);
    //printf("\n");
    //float epr = nnztotal / (300.00 * 784.00);
    //printf("NNZ �ٷֱȣ�%f ,", epr);
    //printf("\n");

    float* GPU_CsrValB;
    int* GPU_csrRowPtrB;
    int* GPU_csrColIndB;

    hipMalloc((void**)&GPU_CsrValB, sizeof(float) * nnztotal);
    hipMalloc((void**)&GPU_csrColIndB, sizeof(int) * nnztotal);
    hipMalloc((void**)&GPU_csrRowPtrB, sizeof(int) * (300 + 1));


    hipsparseSdense2csr(handle,
        300,
        784,
        matb,
        GPU_cusparse_B,
        300,
        GPU_nnzperrow,
        GPU_CsrValB,
        GPU_csrRowPtrB,
        GPU_csrColIndB);



    float* GPU_cusparse_A_first_row;
    float* GPU_cusparse_resu;
    //float* cusparse_resu = (float*)malloc(sizeof(float) * 300);


    hipMalloc((void**)&GPU_cusparse_A_first_row, sizeof(float) * 784);
    hipMalloc((void**)&GPU_cusparse_resu, sizeof(float) * 300);

    hipMemcpy(GPU_cusparse_A_first_row, A_first_row, sizeof(float) * 784, hipMemcpyHostToDevice);

    float alpha = 1;
    float beta = 0;
    HIPSPARSE_OPERATION_NON_TRANSPOSE;

    //double td1 = get_time();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    hipsparseScsrmv(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        300,
        784,
        nnztotal,
        &alpha,
        matb,
        GPU_CsrValB,
        GPU_csrRowPtrB,
        GPU_csrColIndB,
        GPU_cusparse_A_first_row,
        &beta,
        GPU_cusparse_resu);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //hipDeviceSynchronize();
   // double td2 = get_time();
    //td1 = td2 - td1;

    printf("CSR ����ʱ�䣺%f S", time);
    //printf("\n");

    //hipMemcpy(cusparse_resu, GPU_cusparse_resu, sizeof(float) * 300, hipMemcpyDeviceToHost);

    free(clo_B1_dim1);
    free(nnzperrow);
    hipFree(GPU_cusparse_B);
    hipFree(GPU_nnzperrow);
    hipFree(GPU_CsrValB);
    hipFree(GPU_csrColIndB);
    hipFree(GPU_csrRowPtrB);
    hipFree(GPU_cusparse_A_first_row);
    hipFree(GPU_cusparse_resu);


}





void BSR_MV(float** B, float* ROW, int SIZE)


{
    /*---------------����nnzb----------------------*/
    float** BSR_B = (float**)malloc(sizeof(float*) * 304);
    for (int i = 0; i < 304; i++)
    {
        BSR_B[i] = (float*)malloc(sizeof(float) * 784);

    }

    for (int i = 0; i < 304; i++)
        for (int j = 0; j < 784; j++)
            BSR_B[i][j] = 0;


    for (int i = 0; i < 300; i++)
        for (int j = 0; j < 784; j++)
            BSR_B[i][j] = B[i][j];


    int nnzb1 = 0;
    int Block_size = 4;
    int heart_row = 0;
    int heart_clo = 0;
    int flag = 0;
    for (int i = 0; i < 304 / Block_size; i++)
        for (int j = 0; j < 784 / Block_size; j++)
        {
            heart_row = i * Block_size;
            heart_clo = j * Block_size;

            flag = 0;//0��ʾ����û�з���Ԫ
            for (int i1 = 0; i1 < Block_size; i1++)
                for (int j1 = 0; j1 < Block_size; j1++)

                {
                    //printf("(%d, %d)\n",heart_row,heart_clo);
                    if (BSR_B[heart_row + i1][heart_clo + j1] != 0)
                        flag = 1;


                }
            if (flag == 1)
                nnzb1 = nnzb1 + 1;


        }


    printf(" nnzb1��%d\n", nnzb1);


    /*----------------------�ȼ���csr��ʽ------------------------------*/

    float* clo_B1_dim1 = (float*)malloc(sizeof(float) * 784 * 304);
    for (int i = 0; i < 784; i++)
        for (int j = 0; j < 304; j++)
        {
            clo_B1_dim1[i * 304 + j] = BSR_B[j][i];


        }//304*784 ��չ��


    int* nnzperrow = (int*)malloc(sizeof(int) * 304);

    int* GPU_nnzperrow;

    int nnztotal;

    float* GPU_cusparse_B;
    hipMalloc((void**)&GPU_cusparse_B, sizeof(float) * 304 * 784);
    hipMalloc((void**)&GPU_nnzperrow, sizeof(int) * 304);

    hipsparseHandle_t     handle = 0;//�������
    hipsparseMatDescr_t matb = 0;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&matb);
    hipsparseSetMatType(matb, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matb, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t matc = 0;
    hipsparseCreateMatDescr(&matc);
    hipsparseSetMatType(matc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matc, HIPSPARSE_INDEX_BASE_ZERO);


    hipMemcpy(GPU_cusparse_B, clo_B1_dim1, sizeof(float) * 304 * 784, hipMemcpyHostToDevice);

    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, 304, 784, matb, GPU_cusparse_B, 304, GPU_nnzperrow, &nnztotal);//����ÿһ�з���Ԫ


    hipMemcpy(nnzperrow, GPU_nnzperrow, sizeof(int) * 304, hipMemcpyDeviceToHost);

    // printf("NNZ ������%d ,", nnztotal);
    //  printf("\n");
     // float epr = nnztotal / (300.00 * 784.00);
     // printf("NNZ �ٷֱȣ�%f ,", epr);
     // printf("\n");

    float* GPU_CsrValB;
    int* GPU_csrRowPtrB;
    int* GPU_csrColIndB;

    hipMalloc((void**)&GPU_CsrValB, sizeof(float) * nnztotal);
    hipMalloc((void**)&GPU_csrColIndB, sizeof(int) * nnztotal);
    hipMalloc((void**)&GPU_csrRowPtrB, sizeof(int) * (304 + 1));


    hipsparseSdense2csr(handle,
        304,
        784,
        matb,
        GPU_cusparse_B,
        304,
        GPU_nnzperrow,
        GPU_CsrValB,
        GPU_csrRowPtrB,
        GPU_csrColIndB);


    /*----------------------------crsתbsr------------------------*/

    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
    int base, nnzb;
    int m = 304;
    int n = 784;
    int rowBlockDim = SIZE;
    int colBlockDim = SIZE;
    int mb = (m + rowBlockDim - 1) / rowBlockDim;
    int nb = (n + colBlockDim - 1) / colBlockDim;
    int bufferSize;
    void* pBuffer;
    int* bsrRowPtrC;

    hipsparseScsr2gebsr_bufferSize(handle, dir, m, n,
        matb, GPU_CsrValB, GPU_csrRowPtrB, GPU_csrColIndB,
        rowBlockDim, colBlockDim,
        &bufferSize);

    hipMalloc((void**)&pBuffer, bufferSize);
    hipMalloc((void**)&bsrRowPtrC, sizeof(int) * (mb + 1));

    int* nnzTotalDevHostPtr = &nnzb;
    hipsparseXcsr2gebsrNnz(handle, dir, m, n,
        matb, GPU_csrRowPtrB, GPU_csrColIndB,
        matc, bsrRowPtrC, rowBlockDim, colBlockDim,
        nnzTotalDevHostPtr,
        pBuffer);

    if (NULL != nnzTotalDevHostPtr) {
        nnzb = *nnzTotalDevHostPtr;
    }
    else {
        hipMemcpy(&nnzb, bsrRowPtrC + mb, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&base, bsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzb -= base;
    }
    // printf("nnz_per:%f\n", nnzb*SIZE*SIZE/(float)(784*304));

    printf("nnzb2  %d\n", nnzb);
    int* bsrColIndC;
    float* bsrValC;
    hipMalloc((void**)&bsrColIndC, sizeof(int) * nnzb);
    hipMalloc((void**)&bsrValC, sizeof(float) * (rowBlockDim * colBlockDim) * nnzb);
    hipsparseScsr2gebsr(handle, dir, m, n,
        matb,
        GPU_CsrValB, GPU_csrRowPtrB, GPU_csrColIndB,
        matc,
        bsrValC, bsrRowPtrC, bsrColIndC,
        rowBlockDim, colBlockDim,
        pBuffer);


    /*---------------------BSR ���-----------------------*/
    float* x;
    float* GPU_BSR_resu;

    hipMalloc((void**)&x, sizeof(float) * 784);
    hipMalloc((void**)&GPU_BSR_resu, sizeof(float) * 304);
    hipMemcpy(x, ROW, sizeof(float) * 784, hipMemcpyHostToDevice);
    // hipMemcpy(, hy, sizeof(float) * m, hipMemcpyHostToDevice);

    float alpha = 1.0;
    float beta = 0;



    hipsparseSbsrmv(handle, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha,
        matc, bsrValC, bsrRowPtrC, bsrColIndC, SIZE, x, &beta, GPU_BSR_resu);
    hipEvent_t start, stop;
    float time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    hipsparseSbsrmv(handle, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha,
        matc, bsrValC, bsrRowPtrC, bsrColIndC, SIZE, x, &beta, GPU_BSR_resu);


    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("����ʱ�䣺%f\n", time);



    float* BSR_resu = (float*)malloc(sizeof(float) * 304);

    hipMemcpy(BSR_resu, GPU_BSR_resu, sizeof(float) * 304, hipMemcpyDeviceToHost);
    // for (int i = 0; i < 304; i++)
       //  printf("%f, ", BSR_resu[i]);
    // printf("\n");


      /*-------------�ͷſռ�--------------------*/


    hipFree(GPU_cusparse_B);
    hipFree(GPU_nnzperrow);
    hipFree(GPU_CsrValB);
    hipFree(GPU_csrColIndB);
    hipFree(GPU_csrRowPtrB);
    hipFree(pBuffer);
    hipFree(bsrRowPtrC);
    hipFree(pBuffer);
    hipFree(bsrRowPtrC);

    free(BSR_resu);
    for (int i = 0; i < 304; i++)
        free(BSR_B[i]);
    free(BSR_B);
    free(clo_B1_dim1);

}

void BSR_MV_Large(float** B, float* ROW, int SIZE, int how_times)


{
    /*---------------����nnzb----------------------*/
    float** BSR_B = (float**)malloc(sizeof(float*) * 304);
    for (int i = 0; i < 304; i++)
    {
        BSR_B[i] = (float*)malloc(sizeof(float) * 784 * how_times);

    }

    for (int i = 0; i < 304; i++)
        for (int j = 0; j < 784 * how_times; j++)
            BSR_B[i][j] = 0;

    for (int k = 0; k < how_times; k++)
        for (int i = 0; i < 300; i++)
            for (int j = 0; j < 784; j++)
                BSR_B[i][k * 784 + j] = B[i][j];


    int nnzb1 = 0;
    int Block_size = SIZE;
    int heart_row = 0;
    int heart_clo = 0;
    int flag = 0;
    /* for (int i = 0; i < 304 / Block_size; i++)
         for (int j = 0; j < 784 / Block_size; j++)
         {
             heart_row = i * Block_size;
             heart_clo = j * Block_size;

             flag = 0;//0��ʾ����û�з���Ԫ
             for (int i1 = 0; i1 < Block_size; i1++)
                 for (int j1 = 0; j1 < Block_size; j1++)

                 {
                     //printf("(%d, %d)\n",heart_row,heart_clo);
                     if (BSR_B[heart_row + i1][heart_clo + j1] != 0)
                         flag = 1;


                 }
             if (flag == 1)
                 nnzb1 = nnzb1 + 1;


         }


     printf(" nnzb1��%d\n", nnzb1);

     */
     /*----------------------�ȼ���csr��ʽ------------------------------*/

    float* clo_B1_dim1 = (float*)malloc(sizeof(float) * 784 * how_times * 304);
    for (int k = 0; k < how_times; k++)
        for (int i = 0; i < 784; i++)
            for (int j = 0; j < 304; j++)
            {
                clo_B1_dim1[k * 784 * 304 + i * 304 + j] = BSR_B[j][k * 784 + i];


            }//how_timeS*304*784 ��չ��


    int* nnzperrow = (int*)malloc(sizeof(int) * 304);

    int* GPU_nnzperrow;

    int nnztotal;

    float* GPU_cusparse_B;
    hipMalloc((void**)&GPU_cusparse_B, sizeof(float) * 304 * 784 * how_times);
    hipMalloc((void**)&GPU_nnzperrow, sizeof(int) * 304);

    hipsparseHandle_t     handle = 0;//�������
    hipsparseMatDescr_t matb = 0;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&matb);
    hipsparseSetMatType(matb, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matb, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t matc = 0;
    hipsparseCreateMatDescr(&matc);
    hipsparseSetMatType(matc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matc, HIPSPARSE_INDEX_BASE_ZERO);


    hipMemcpy(GPU_cusparse_B, clo_B1_dim1, sizeof(float) * 304 * 784 * how_times, hipMemcpyHostToDevice);

    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, 304, 784 * how_times, matb, GPU_cusparse_B, 304, GPU_nnzperrow, &nnztotal);//����ÿһ�з���Ԫ


    hipMemcpy(nnzperrow, GPU_nnzperrow, sizeof(int) * 304, hipMemcpyDeviceToHost);

    // printf("NNZ ������%d ,", nnztotal);
    //  printf("\n");
     // float epr = nnztotal / (300.00 * 784.00);
     // printf("NNZ �ٷֱȣ�%f ,", epr);
     // printf("\n");

    float* GPU_CsrValB;
    int* GPU_csrRowPtrB;
    int* GPU_csrColIndB;

    hipMalloc((void**)&GPU_CsrValB, sizeof(float) * nnztotal);
    hipMalloc((void**)&GPU_csrColIndB, sizeof(int) * nnztotal);
    hipMalloc((void**)&GPU_csrRowPtrB, sizeof(int) * (304 + 1));


    hipsparseSdense2csr(handle,
        304,
        784 * how_times,
        matb,
        GPU_cusparse_B,
        304,
        GPU_nnzperrow,
        GPU_CsrValB,
        GPU_csrRowPtrB,
        GPU_csrColIndB);


    /*----------------------------crsתbsr------------------------*/

    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
    int base, nnzb;
    int m = 304;
    int n = 784 * how_times;
    int rowBlockDim = SIZE;
    int colBlockDim = SIZE;
    int mb = (m + rowBlockDim - 1) / rowBlockDim;
    int nb = (n + colBlockDim - 1) / colBlockDim;
    int bufferSize;
    void* pBuffer;
    int* bsrRowPtrC;

    hipsparseScsr2gebsr_bufferSize(handle, dir, m, n,
        matb, GPU_CsrValB, GPU_csrRowPtrB, GPU_csrColIndB,
        rowBlockDim, colBlockDim,
        &bufferSize);

    hipMalloc((void**)&pBuffer, bufferSize);
    hipMalloc((void**)&bsrRowPtrC, sizeof(int) * (mb + 1));

    int* nnzTotalDevHostPtr = &nnzb;
    hipsparseXcsr2gebsrNnz(handle, dir, m, n,
        matb, GPU_csrRowPtrB, GPU_csrColIndB,
        matc, bsrRowPtrC, rowBlockDim, colBlockDim,
        nnzTotalDevHostPtr,
        pBuffer);

    if (NULL != nnzTotalDevHostPtr) {
        nnzb = *nnzTotalDevHostPtr;
    }
    else {
        hipMemcpy(&nnzb, bsrRowPtrC + mb, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&base, bsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzb -= base;
    }
    // printf("nnz_per:%f\n", nnzb * SIZE * SIZE / (float)(784*how_times * 304));
     printf("nnzb:%d\n", nnzb);
    int* bsrColIndC;
    float* bsrValC;
    hipMalloc((void**)&bsrColIndC, sizeof(int) * nnzb);
    hipMalloc((void**)&bsrValC, sizeof(float) * (rowBlockDim * colBlockDim) * nnzb);
    hipsparseScsr2gebsr(handle, dir, m, n,
        matb,
        GPU_CsrValB, GPU_csrRowPtrB, GPU_csrColIndB,
        matc,
        bsrValC, bsrRowPtrC, bsrColIndC,
        rowBlockDim, colBlockDim,
        pBuffer);


    /*---------------------BSR ���-----------------------*/
    float* Large_Row = (float*)malloc(sizeof(float*) * 784 * how_times);
    for (int i = 0; i < how_times; i++)
        for (int j = 0; j < 784; j++)
            Large_Row[i * 784 + j] = ROW[j];


    float* x;
    float* GPU_BSR_resu;

    hipMalloc((void**)&x, sizeof(float) * 784 * how_times);
    hipMalloc((void**)&GPU_BSR_resu, sizeof(float) * 304);
    hipMemcpy(x, Large_Row, sizeof(float) * 784 * how_times, hipMemcpyHostToDevice);
    // hipMemcpy(, hy, sizeof(float) * m, hipMemcpyHostToDevice);

    float alpha = 1.0;
    float beta = 0;



    hipsparseSbsrmv(handle, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha,
        matc, bsrValC, bsrRowPtrC, bsrColIndC, SIZE, x, &beta, GPU_BSR_resu);
    hipEvent_t start, stop;
    float time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    hipsparseSbsrmv(handle, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha,
        matc, bsrValC, bsrRowPtrC, bsrColIndC, SIZE, x, &beta, GPU_BSR_resu);


    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("BSR���СΪ%d����ʱ�䣺%f S\n", Block_size, time);



    float* BSR_resu = (float*)malloc(sizeof(float) * 304);

    hipMemcpy(BSR_resu, GPU_BSR_resu, sizeof(float) * 304, hipMemcpyDeviceToHost);
    // for (int i = 0; i < 304; i++)
        // printf("%f, ", BSR_resu[i]);
     //printf("\n");


      /*-------------�ͷſռ�--------------------*/


    hipFree(GPU_cusparse_B);
    hipFree(GPU_nnzperrow);
    hipFree(GPU_CsrValB);
    hipFree(GPU_csrColIndB);
    hipFree(GPU_csrRowPtrB);
    hipFree(pBuffer);
    hipFree(bsrRowPtrC);
    hipFree(pBuffer);
    hipFree(bsrRowPtrC);

    free(BSR_resu);
    for (int i = 0; i < 304; i++)
        free(BSR_B[i]);
    free(BSR_B);
    free(clo_B1_dim1);

}

void get_block_matrix2(float** A, int m, int n, int Block_size, int percent)//����A����Ϊ304*784
{
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            A[i][j] = 0;//����

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            //printf("%f, ", A[i][j]);
        }
        // printf("\n");

    }

    //  printf("\n");


    int Row_size = m / Block_size;
    int Clo_size = n / Block_size;

    // printf("%d\n", Row_size);
    // printf("%d\n", Clo_size);


    int block_nnzNum = n * m * percent / (100 * Block_size * Block_size);
    // printf("����%d\n", block_nnzNum);
    int* row = (int*)malloc(sizeof(int) * block_nnzNum);
    int* clo = (int*)malloc(sizeof(int) * block_nnzNum);
    float* value = (float*)malloc(sizeof(float) * block_nnzNum * Block_size * Block_size);


    //��������Ԫ

    srand((unsigned)time(NULL) + 23);
    for (int i = 0; i < block_nnzNum; i++)
    {
        row[i] = rand() % Row_size;
        // printf("%d, ", row[i]);
    }
    // printf("\n");

    srand((unsigned)time(NULL) + 43);
    for (int i = 0; i < block_nnzNum; i++)
    {
        clo[i] = rand() % Clo_size;
        //printf("%d, ", clo[i]);
    }
    //printf("\n");

    srand((unsigned)time(NULL) + 67);
    for (int i = 0; i < block_nnzNum * Block_size * Block_size; i++)
    {
        value[i] = (rand() % 10000) / 100.0 + 1.0;
        // printf("%f, ", value[i]);
    }
    // printf("\n");
    for (int i = 0; i < block_nnzNum; i++)
    {

        for (int j = 0; j < Block_size; j++)
            for (int k = 0; k < Block_size; k++)
                A[row[i] * Block_size + j][clo[i] * Block_size + k] = value[i * Block_size * Block_size + j * Block_size + k];





    }

    int nnz = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (A[i][j] != 0)
                nnz = nnz + 1;
        }


    }

    float per = nnz / (float)(m * n);
    printf("����Ԫ��ռ�ٷֱȣ�%f\n", per);
    // printf("nnz��%d\n", nnz);
     //printf("nnzb1��%d\n", nnz/(Block_size*Block_size));
   // for (int i = 0; i < m; i++)
    //{
      //  for (int j = 0; j < n; j++)
        //{
          //  printf("%f,", A[i][j]);
        //}
        //printf("\n");

    //}

    //printf("\n");

    printf("���ɿ��СΪ%d�ľ���", Block_size);


}


void blas_mv(float** B, float* A_first_row)
{

    float* clo_B1_dim1 = (float*)malloc(sizeof(float) * 784 * 304);
    for (int i = 0; i < 784; i++)
        for (int j = 0; j < 304; j++)
        {
            clo_B1_dim1[i * 304 + j] = B[j][i];


        }//304*784 ��չ��




    int ida1 = 304;
    float alf = 1;
    float bet = 0;
    float* alpha1 = &alf;
    float* beta1 = &bet;
    float* cublas_resu = (float*)malloc(sizeof(float) * 304);
    float* GPU_cublas_resu;
    float* GPU_cusparse_B;
    float* GPU_cusparse_A_first_row;

    hipMalloc((void**)&GPU_cublas_resu, sizeof(float) * 304);
    hipMalloc((void**)&GPU_cusparse_B, sizeof(float) * 304 * 784);
    hipMalloc((void**)&GPU_cusparse_A_first_row, sizeof(float) * 784);


    hipMemcpy(GPU_cusparse_B, clo_B1_dim1, sizeof(float) * 304 * 784, hipMemcpyHostToDevice);
    hipMemcpy(GPU_cusparse_A_first_row, A_first_row, sizeof(float) * 784, hipMemcpyHostToDevice);

    hipblasHandle_t handle1;
    hipblasCreate(&handle1);

    hipblasSgemv(handle1, HIPBLAS_OP_N,
        304, 784,
        alpha1,
        GPU_cusparse_B, ida1,
        GPU_cusparse_A_first_row, 1,
        beta1,
        GPU_cublas_resu, 1);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    float time = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);


    hipblasSgemv(handle1, HIPBLAS_OP_N,
        304, 784,
        alpha1,
        GPU_cusparse_B, ida1,
        GPU_cusparse_A_first_row, 1,
        beta1,
        GPU_cublas_resu, 1);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    // hipDeviceSynchronize();
     //double td2 = get_time();
     //td1 = td2 - td1;

    hipEventDestroy(stop);
    hipEventDestroy(start);
    printf("cublas ����ʱ�䣺%f", time);
    printf("\n");


    hipMemcpy(cublas_resu, GPU_cublas_resu, sizeof(float) * 304, hipMemcpyDeviceToHost);

    for (int i = 0; i < 304; i++)
        printf("%f, ", cublas_resu[i]);
    printf("\n");



    free(clo_B1_dim1);


}
int main(int argc, char* argv[]) {



    float** rand_sparse;
    rand_sparse = (float**)malloc(sizeof(float*) * 300);
    for (int i = 0; i < 300; i++)
    {
        rand_sparse[i] = (float*)malloc(sizeof(float) * 784);

    }


   


    //�����ļ�picture

    float** A;
    A = (float**)malloc(sizeof(float*) * 50);
    for (int i = 0; i < 50; i++)
    {
        A[i] = (float*)malloc(sizeof(float) * 784);

    }

    FILE* fw = fopen("picture.txt", "r");

    for (int i = 0; i < 50; i++)
    {
        for (int j = 0; j < 784; j++)
        {
            fscanf(fw, "%f", &A[i][j]);




        }




    }

    fclose(fw);
    //ȡA����ĵ�һ��
    float* A_first_row = (float*)malloc(sizeof(float) * 784);
    for (int i = 0; i < 784; i++)
        A_first_row[i] = A[0][i];

    //�����ļ�layer1

    float** B1;
    B1 = (float**)malloc(sizeof(float*) * 300);
    for (int i = 0; i < 300; i++)
    {
        B1[i] = (float*)malloc(sizeof(float) * 784);

    }

    float** C;

    /*C = (float**)malloc(sizeof(float*) * 50);
    for (int i = 0; i < 50; i++)
    {
        C[i] = (float*)malloc(sizeof(float) * 300);

    }*/


    FILE* fw2 = fopen("layer1.txt", "r");

    for (int i = 0; i < 300; i++)
    {
        for (int j = 0; j < 784; j++)
        {
            fscanf(fw, "%f", &B1[i][j]);




        }




    }

    fclose(fw2);

    //����B���󣬽�B1ת��

    float** B;
    B = (float**)malloc(sizeof(float*) * 784);
    for (int i = 0; i < 784; i++)
    {
        B[i] = (float*)malloc(sizeof(float) * 300);

    }
    for (int i = 0; i < 300; i++)
    {
        for (int j = 0; j < 784; j++)
        {
            B[j][i] = B1[i][j];




        }




    }

    float** B_dense;
    B_dense = (float**)malloc(sizeof(float*) * 304);
    for (int i = 0; i < 304; i++)
    {
        B_dense[i] = (float*)malloc(sizeof(float) * 784);

    }
    float** B_sparse;
    B_sparse = (float**)malloc(sizeof(float*) * 304);
    for (int i = 0; i < 304; i++)
    {
        B_sparse[i] = (float*)malloc(sizeof(float) * 784);

    }
    float** B_ran;
    B_ran = (float**)malloc(sizeof(float*) * 304);
    for (int i = 0; i < 304; i++)
    {
        B_ran[i] = (float*)malloc(sizeof(float) * 784);

    }


    // get_block(B1, B_dense, B_sparse);



    // BSR_MV_Large(B_dense, A_first_row,16, 100);


     get_block_matrix2(B_ran,304,784, 4, 19);
     printf("\n");
     printf("----------------------------------\n");
     BSR_MV_Large(B_ran, A_first_row, 1, 20);
    // BSR_MV_Large(B_ran, A_first_row, 2, 10);
     BSR_MV_Large(B_ran, A_first_row, 4, 20);
    // BSR_MV_Large(B_ran, A_first_row, 8, 1);
    // crsmv(B_ran, A_first_row);
     printf("\n");
     printf("----------------------------------\n");
   //  printf("---------------blas-------------------\n");
    // blas_mv(B_ran, A_first_row);
    // printf("----------------------------------\n");

   
 
  


    for (int i = 0; i < 784; i++)
        free(B[i]);
    free(B);
    for (int i = 0; i < 300; i++)
        free(B1[i]);
    free(B1);
    for (int i = 0; i < 50; i++)
        free(A[i]);
    free(A);
    for (int i = 0; i < 300; i++)
        free(rand_sparse[i]);
    free(rand_sparse);
    for (int i = 0; i < 304; i++)
        free(B_dense[i]);
    free(B_dense);
    for (int i = 0; i < 304; i++)
        free(B_ran[i]);
    free(B_ran);
    for (int i = 0; i < 304; i++)
        free(B_sparse[i]);
    free(B_sparse);
   






}